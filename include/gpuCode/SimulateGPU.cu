#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "../transportbase.h"

// ASSISTANT KERNELS //////////////////////////////////////////////////////////

// The generator is used for creating pseudo-random numbers for a given array
// of states (std_normal distribution)
__device__ float generateNormal(hiprandState* globalState, const unsigned int
        ind) {
    //copy state to local mem
    hiprandState localState = globalState[ind];
    //apply uniform distribution with calculated random
    float rndval = hiprand_normal( &localState );
    //update state
    globalState[ind] = localState;
    //return value
    return rndval;
}

// Initialise the states for hiprand on each kernel
__global__ void initialise_curand_on_kernels(hiprandState* state,
        unsigned long seed) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

// Generate the random numbers required for use in another function
__global__ void set_random_number_from_kernels(float* _ptr, hiprandState*
        globalState, const unsigned int _points, const unsigned int
        dimension1, const unsigned int dimension2 = 1, const unsigned int
        dimension3 = 1) {

    // Get the global index for the matrix
    unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;

    // Each generator operates on a different set of nYears * nPaths
    unsigned int stateNo = (unsigned int)(idx / (dimension2*dimension3));

    //only call gen on the kernels we have inited
    //(one per device container element)
    if (stateNo < dimension1) {
        if (idx < _points)
        {
            _ptr[idx] = generateNormal(&globalState[stateNo], idx);
        }
    }
}

// MAIN KERNELS ///////////////////////////////////////////////////////////////

// Kernel for computing a single path of an uncertain variable
__global__ void expPVPath(const int noPaths, const float gr, const int nYears,
        const float meanP, const float timeStep, const float rrr, float
        current, float reversion, float jumpProb, const float* brownian, const
        float* jumpSize, const float* jump, float* result) {

    // Get the global index for the matrix
    unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;

    if (idx < noPaths) {
        // Simulate a forward path
        float value = 0;
        float curr = current;

        for (int ii = 0; ii < nYears; ii++) {
            float jumped = (jump[idx+ii] < jumpProb)? 1.0f : 0.0f;

            curr += reversion*(meanP - curr)*timeStep + curr*brownian[idx+ii] +
                    exp(jumpSize[idx+ii] - 1)*curr*jumped;
            value += pow(1 + gr,ii)*curr/pow((1 + rrr),ii);
        }

        result[idx] = value;
    }
}

// The matrix multiplication kernel parallelises the multiplication of Eigen
// matrices
__global__ void matrixMultiplicationKernelNaive(const float* A, const float* B,
        float* C, int a, int b, int c, int d) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0.0f;

    if (ROW < a && COL < d) {
        // each thread computes one element of the block sub-matrix
        for (int ii = 0; ii < b; ii++) {
            tmpSum += A[ROW * b + ii] * B[ii * b + COL];
        }
    }
    C[ROW * a + COL] = tmpSum;
}

// The matrix element-wise multiplication kernel parallelises the element-wise
// multiplication of Eigen matrices

__global__ void matrixElementWiseMultiplicationKernelNaive(const float* A,
        const float* B, float* C, int a, int b) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    if (ROW < a && COL < b) {
        C[ROW * a + COL] = A[ROW * b + COL]*B[ROW * b + COL];
    }
}

// The optimised matrix multiplication kernel that relies on efficient memory
// management
__global__ void matrixMultiplicationKernel(float *A, float* B, float* C, int a,
        int b, int d) {

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int ROW = by*blockDim.y+ty;
    int COL = bx*blockDim.x+tx;

    // First check if the thread exceeds the matrix dimensions
    if (ROW < a && COL < d) {

        // Declaration of the shared memory array As used to store the sub-
        // matrix of A
        __shared__ float As[BLOCK_SIZE * BLOCK_SIZE];
        __shared__ float As2[BLOCK_SIZE * BLOCK_SIZE];

        float *prefetch = As;
        float *prefetch2 = As2;

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        // __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        float cv[BLOCK_SIZE];

        for (int ii = 0; ii < BLOCK_SIZE; ii++) {
             cv[ii] = 0;
        }

        // Index of the first sub-matrix of A processed by the block
        int aBegin = a * BLOCK_SIZE * by;

        // Index of the last sub-matrix of A processed by the block
        int aEnd   = aBegin + a - 1;

        // Step size used to iterate through the sub-matrices of A
        int aStep  = BLOCK_SIZE;

        // Index of the first sub-matrix of B processed by the block
        int bBegin = BLOCK_SIZE * VECTOR_SIZE * bx;

        // Step size used to iterate through the sub-matrices of B
        int bStep  = BLOCK_SIZE * d;

        int cBegin = d * BLOCK_SIZE * by + VECTOR_SIZE * BLOCK_SIZE * bx;

        // Csub is used to store the element of the block sub-matrix
        // that is computed by the thread
        // float Csub = 0;
        float *Ap = &A[aBegin + a * ty +tx];
        float *ap = &prefetch[ty + BLOCK_SIZE * tx];
#pragma unroll
        for(int ii = 0; ii < BLOCK_SIZE; ii+=4){
          ap[ii] = Ap[a * ii];
        }
        __syncthreads();

        // Loop over all the sub-matrices of A and B
        // required to compute the block sub-matrix
        for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) {

            // Load the matrices from device memory
            // to shared memory; each thread loads
            // one element of each matrix
            Ap = &A[a + aStep + a * ty +tx];
            float *ap2 = &prefetch2[ty + BLOCK_SIZE * tx];
#pragma unroll
            for(int ii = 0; ii < BLOCK_SIZE; ii+=4){
                ap2[ii] = Ap[b * ii];
            }

            ap = &prefetch[0];
            float *bp = &B[b + BLOCK_SIZE * ty + tx];

#pragma unroll
            for (int ii = 0; ii < BLOCK_SIZE; ii++) {
                float bv = bp[0];
                for (int jj = 0; jj < BLOCK_SIZE; jj++) {
                    cv[jj] += ap[jj]*bv;
                    ap += BLOCK_SIZE;
                    bp += d;
                }
            }

            // Synchronize to make sure the matrices are loaded
            __syncthreads();

            // swap As and As2
            float *prefetch_temp = prefetch;
            prefetch = prefetch2;
            prefetch2 = prefetch_temp;
        }

        // Write the block sub-matrix to device memory;
        // each thread writes one element
        float *Cp = &C[cBegin];
        Cp += BLOCK_SIZE * ty + tx;
        int cStep = d;
#pragma unroll
        for(int ii=0; ii<BLOCK_SIZE; ii++){
          Cp[0] = cv[ii]; Cp += cStep;
        }
    }
}

// Element-wise matrix multiplication kernel
__global__ void matrixMultiplicationKernelEW(const float* A, const float*
        B, float* C, int a, int b) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    if (ROW < a && COL < b) {
        C[ROW * a + COL] = A[ROW * b + COL]*B[ROW * b + COL];
    }
}

// Element-wise matrix division kernel
__global__ void matrixDivisionKernelEW(const float* A, const float* B,
        float* C, int a, int b) {

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    if (ROW < a && COL < b) {
        C[ROW * a + COL] = A[ROW * b + COL]/B[ROW * b + COL];
    }
}

// Computes whether there is an intersection between line segements or not
__global__ void pathAdjacencyKernel(int noTransitions, int noSegments,
        float* XY1, float* XY2, float* X4_X3, float* Y4_Y3, float* X2_X1,
        float* Y2_Y1, int* adjacency) {

    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int idx = blockId * blockDim.x + threadIdx.x;

    if (idx < noTransitions*noSegments) {
        int seg1 = idx/noSegments;
        int seg2 = idx - seg1*noSegments;

        float Y1_Y3 = XY1[seg1 + noTransitions] - XY2[seg2 + noSegments];
        float X1_X3 = XY1[seg1] - XY2[seg2];

        float numa = X4_X3[seg2]*Y1_Y3 - Y4_Y3[seg2]*X1_X3;
        float numb = X2_X1[seg1]*Y1_Y3 - Y2_Y1[seg1]*X1_X3;
        float deno = Y4_Y3[seg2]*X2_X1[seg1] - X4_X3[seg2]*Y2_Y1[seg1];

        float u_a = numa/deno;
        float u_b = numb/deno;

        adjacency[idx] = (int)((u_a >= 0.0) && (u_a <= 1.0) && (u_b >= 0.0)
                && (u_b <= 1.0));
    }
}

// Sums the line segments intersection values along the each row
__global__ void roadCrossingsKernel(int rows, int segs, int* adjacency,
        int* cross) {

    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < rows) {
        cross[idx] = 0;

        for (int ii = 0; ii < segs; ii++) {
            cross[idx] += adjacency[idx*segs + ii];
        }
    }
}

// The patch kernel represents a single cell for generating habitat patches
// The results matrix contains the following:
//
__global__ void patchComputation(int noCandidates, int W, int H, int skpx, int
        skpy, int xres, int yres, float subPatchArea, float xspacing, float
        yspacing, float capacity, int uniqueRegions, const int* labelledImage,
        const float* pops, float* results) {

    // Get global index of thread
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < noCandidates) {
        // Get large grid cell subscripts of thread
        int blockIdxY = (int)(((int)(idx/uniqueRegions))/xres);
        int blockIdxX = (int)(idx/uniqueRegions) - blockIdxY*xres;
        // Valid region numbering starts at 1, not 0
        int regionNo = idx - blockIdxY*xres*uniqueRegions - blockIdxX*
                uniqueRegions + 1;

        int blockSizeX;
        int blockSizeY;

        if ((blockIdxX+1)*skpx <= H) {
            blockSizeX = skpx;
        } else {
            blockSizeX = H-blockIdxX*skpx;
        }

        if ((blockIdxY+1)*skpy <= W) {
            blockSizeY = skpy;
        } else {
            blockSizeY = W-blockIdxY*skpy;
        }

        // Iterate through each sub patch for this large grid cell
        float area = 0.0f;
        float cap = 0.0f;
        float pop = 0.0f;
        float cx = 0.0f;
        float cy = 0.0f;

        for (int ii = 0; ii < blockSizeX; ii++) {
            for (int jj = 0; jj < blockSizeY; jj++) {
                int subIdx = blockIdxY*xres*skpx*skpy + blockIdxX*skpx
                        + jj*H + ii;
                area += (float)(labelledImage[subIdx] == regionNo);
            }
        }

        if (area > 0) {
            for (int ii = 0; ii < blockSizeX; ii++) {
                for (int jj = 0; jj < blockSizeY; jj++) {
                    int subIdx = blockIdxY*xres*skpx*skpy + blockIdxX*skpx
                            + jj*H + ii;
                    pop += pops[subIdx];
                    cx += ii*(float)(labelledImage[subIdx] == regionNo);
                    cy += jj*(float)(labelledImage[subIdx] == regionNo);
                }
            }
            cx = xspacing*(cx/area + blockIdxX*skpx);
            cy = yspacing*(cy/area + blockIdxY*skpy);
            area = area*subPatchArea;
            cap = area*capacity;
        }

        // Store results to output matrix
        results[5*idx] = area;
        results[5*idx+1] = cap;
        results[5*idx+2] = pop;
        results[5*idx+3] = cx;
        results[5*idx+4] = cy;

//        printf("%4d, %5d, %8.0f, %5.0f, %5.0f, %5.0f, %5.0f\n",idx,blockSizeX,
//                results[5*idx],results[5*idx+1],results[5*idx+2],
//                results[5*idx+3],results[5*idx+4]);
    }
}

// The mte kernel represents a single path for mte
__global__ void mteKernel(int noPaths, int nYears, int noPatches, float grm,
        float grsd, float *initPops, float* caps, float* mmm, float* eps,
        float* drf) {
    // Global index for finding the thread number
    int ii = blockIdx.x*blockDim.x + threadIdx.x;

    // Only perform matrix multiplication sequentially for now. Later, if
    // so desired, we can use dynamic parallelism because the card in the
    // machine has CUDA compute capability 3.5
    if (ii < noPaths) {
        // Initialise the temporary vector
        float *pops;
        pops = (float*)malloc(noPatches*sizeof(float));
        float *popsOld;
        popsOld = (float*)malloc(noPatches*sizeof(float));

        // Initialise the prevailing population vector
        for (int jj = 0; jj < noPatches; jj ++) {
            pops[jj] = 1.0f;
            popsOld[jj] = initPops[jj];
        }

        for (int jj = 0; jj < nYears; jj++) {
            // Movement and mortality
            for (int kk = 0; kk < noPatches; kk++) {
                pops[kk] = 0.0;
                for (int ll = 0; ll < noPatches; ll++) {
                    pops[kk] += popsOld[ll]*mmm[kk*noPatches+ll];
                }
            }

            // Natural birth and death
            for (int kk = 0; kk < noPatches; kk++) {
                float gr = grsd*drf[ii*(nYears*noPatches) + jj*noPatches + kk]
                        + grm;
                popsOld[kk] = pops[kk]*(1.0f + gr*(caps[kk]-pops[kk])/caps[kk]/
                        100.0);
            }
        }

        eps[ii] = 0.0f;
        for (int jj = 0; jj < noPatches; jj++) {
            eps[ii] += popsOld[jj];
        }
        free(pops);
        free(popsOld);
    }
}

// The kernel for computing forward paths in ROV. This routine does not
// consider
__global__ void forwardPathKernel() {
    // Global thread index
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    // Only perform matrix multiplication sequentially for now. Later, if so
    // desired, we can use dynamic parallelism because the card in the
    // machine has CUDA compute compatability 3.5

    if (idx < nopaths) {
        // Initialise the temporary population vectors
        float *pops;
        pops = (float*)malloc(noPatches*sizeof(float));
        float *popsOld;
        popsOld = (float*)malloc(noPatches*sizeof(float));

        // Initialise the prevailing population vector
        int counter = 0;
        int counter2 = 0;

        for (int ii = 0; ii < noSpecies; ii++) {
            for (int jj = 0; jj < noPatches[ii]; jj++) {
                popsOld[jj + counter] = initPops[jj + counter];
                counter++;
            }
        }

        for (int ii = 0; ii < nYears; ii++) {
            counter = 0;
            counter2 = 0;

            for (int jj = 0; jj < noSpecies; jj++) {
                for (int kk = 0; kk < noPatches[jj]; jj++) {
                    for (int ll = 0; ll < noPatches[jj]; ll++) {
                        // Movement and mortality
                        pops[kk + counter] = popsOld[ll]*transitions[counter2
                                + kk*noPatches[jj] + ll]*survival[counter2
                                + kk*noPatches[jj] + ll];
                        counter2++;
                    }
                    float gr = grsd[jj]*drf[idx*totalPatches*nYears +
                            ii*totalPatches + kk] + grm[jj];
                    popsOld[kk + counter] = pops[kk + counter]*(1.0f + gr*
                            (caps[kk + counter] - pops[kk + counter])/
                            caps[kk + counter]/100.0);
                    counter++;
                }
            }
        }


        for (int ii = 0; ii < nYears; ii++) {
            counter = 0;

            // Populations
            for (int jj = 0; jj < noSpecies; jj++) {
                for (int kk = 0; kk < noPatches[jj]; kk++) {
                    tempPops[] = 0;

                    for (int ll = 0; ll < noPatches[jj]; ll++) {
                        pop +=
                    }
                }
            }

            // Other uncertainties

        }
    }
}

// The rov kernel represents a single path for rov
__global__ void rovKernel() {
      printf("Hello from mykernel\n");
}

// WRAPPERS ///////////////////////////////////////////////////////////////////

void SimulateGPU::expPV(UncertaintyPtr uncertainty) {
    // Get device properties
    int device = 0;
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    maxMultiProcessors = properties.multiProcessorCount;
    maxThreadsPerBlock = properties.maxThreadsPerBlock;

    OptimiserPtr optimiser = uncertainty->getOptimiser();
    EconomicPtr economic = optimiser->getEconomic();
    unsigned int nYears = economic->getYears();
    double timeStep = economic->getTimeStep();
    unsigned int noPaths = optimiser->getOtherInputs()->getNoPaths();
    double total = 0.0;
    double gr = optimiser->getTraffic()->getGR()*economic->getTimeStep();

    // Uncertain components of Brownian motion
    float *d_brownian, *d_jumpSizes, *d_jumps, *d_results, *results;
    hiprandGenerator_t gen;
    srand(time(NULL));
    int _seed = rand();

    results = (float*)malloc(noPaths*sizeof(float));
    hipMalloc((void **)&d_brownian, sizeof(float)*nYears*noPaths);
    hipMalloc((void **)&d_jumpSizes, sizeof(float)*nYears*noPaths);
    hipMalloc((void **)&d_jumps, sizeof(float)*nYears*noPaths);
    hipMalloc((void **)&d_results, sizeof(float)*nYears*noPaths);

    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, _seed);

    hiprandGenerateNormal(gen, d_brownian, nYears*noPaths, 0.0f, uncertainty->
            getNoiseSD()*timeStep);
    hiprandGenerateNormal(gen, d_jumpSizes, nYears*noPaths,
            -pow(uncertainty->getPoissonJump(),2)/2,pow(uncertainty->
            getPoissonJump(),2));
    hiprandGenerateUniform(gen, d_jumps, nYears*noPaths);

    hiprandDestroyGenerator(gen);
    hipDeviceSynchronize();

    // Compute path values
    int noBlocks = (noPaths % maxThreadsPerBlock) ? (int)(
            noPaths/maxThreadsPerBlock + 1) : (int)
            (noPaths/maxThreadsPerBlock);
    int noThreadsPerBlock = min(maxThreadsPerBlock,nYears*noPaths);

    expPVPath<<<noBlocks,noThreadsPerBlock>>>(noPaths, gr, nYears,
            uncertainty->getMean(), timeStep, economic->getRRR(),
            uncertainty->getCurrent(), uncertainty->getMRStrength(),
            uncertainty->getJumpProb(), d_brownian, d_jumpSizes, d_jumps,
            d_results);

    hipDeviceSynchronize();

    hipMemcpy(results,d_results,noPaths*sizeof(float),hipMemcpyDeviceToHost);

    for (int ii = 0; ii < noPaths; ii++) {
        total += results[ii];
    }

    uncertainty->setExpPV((double)total/((double)noPaths));

    total = 0.0;
    for (int ii = 0; ii < noPaths; ii++) {
        total += pow(results[ii] - uncertainty->getExpPV(),2);
    }

    uncertainty->setExpPVSD(sqrt(total));

    hipFree(d_brownian);
    hipFree(d_jumpSizes);
    hipFree(d_jumps);
    hipFree(d_results);
    free(results);
}

void SimulateGPU::eMMN(const Eigen::MatrixXd& A, const Eigen::MatrixXd& B,
        Eigen::MatrixXd& C) {

    int device = 0;
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    maxMultiProcessors = properties.multiProcessorCount;
    maxThreadsPerBlock = properties.maxThreadsPerBlock;

    if (A.cols() != B.rows()) {
        throw "SimulateGPU: matrixMultiplication: Inner dimensions do not match!";
    }

    float *Af, *Bf, *Cf, *d_A, *d_B, *d_C;

    int a = A.rows();
    int b = A.cols();
    int c = B.rows();
    int d = B.cols();

    Af = (float*)malloc(a*b*sizeof(float));
    Bf = (float*)malloc(c*d*sizeof(float));
    Cf = (float*)malloc(a*d*sizeof(float));

    hipMalloc(&d_A,a*b*sizeof(float));
    hipMemcpy(d_A,Af,a*b*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc(&d_B,c*d*sizeof(float));
    hipMemcpy(d_B,Bf,c*d*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc(&d_C,a*d*sizeof(float));

    // declare the number of blocks per grid and the number of threads per block
    dim3 threadsPerBlock(a, d);
    dim3 blocksPerGrid(1, 1);
        if (a*d > maxThreadsPerBlock){
            threadsPerBlock.x = maxThreadsPerBlock;
            threadsPerBlock.y = maxThreadsPerBlock;
            blocksPerGrid.x = ceil(double(a)/double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(d)/double(threadsPerBlock.y));
        }

    matrixMultiplicationKernelNaive<<<blocksPerGrid,threadsPerBlock>>>(d_A,d_B,
            d_C,a,b,c,d);

    // Retrieve result and free data
    hipMemcpy(C.data(),d_C,a*d*sizeof(float),hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    free(Af);
    free(Bf);
    free(Cf);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void SimulateGPU::eMM(const Eigen::MatrixXd& A, const Eigen::MatrixXd& B,
        Eigen::MatrixXd& C) {

    int device = 0;
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    maxMultiProcessors = properties.multiProcessorCount;
    maxThreadsPerBlock = properties.maxThreadsPerBlock;

    if (A.cols() != B.rows()) {
        throw "SimulateGPU: matrixMultiplication: Inner dimensions do not match!";
    }

    float *d_A, *d_B, *d_C;

    int a = A.rows();
    int b = A.cols();
    int c = B.rows();
    int d = B.cols();

    Eigen::MatrixXf Af = A.cast<float>();
    Eigen::MatrixXf Bf = B.cast<float>();
    Eigen::MatrixXf Cf = C.cast<float>();

    hipMalloc(&d_A,a*b*sizeof(float));
    hipMemcpy(d_A,Af.data(),a*b*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc(&d_B,c*d*sizeof(float));
    hipMemcpy(d_B,Bf.data(),c*d*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc(&d_C,a*d*sizeof(float));

    // declare the number of blocks per grid and the number of threads per block
    dim3 threads(BLOCK_SIZE,VECTOR_SIZE);
    dim3 grid(d/(BLOCK_SIZE*VECTOR_SIZE), a/BLOCK_SIZE);

    matrixMultiplicationKernel<<<grid,threads>>>(d_A,d_B,d_C,a,b,d);

    // Retrieve result and free data
    hipMemcpy(Cf.data(),d_C,a*d*sizeof(float),hipMemcpyDeviceToHost);

    C = Cf.cast<double>();

    hipDeviceSynchronize();
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void SimulateGPU::ewMM(const Eigen::MatrixXd& A, const Eigen::MatrixXd &B,
        Eigen::MatrixXd &C) {

    int device = 0;
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    maxMultiProcessors = properties.multiProcessorCount;
    maxThreadsPerBlock = properties.maxThreadsPerBlock;

    if ((A.cols() != B.cols()) || (A.rows() != B.rows())) {
        throw "SimulateGPU: matrixMultiplication: Matrix dimensions do not match!";
    }

    float *d_A, *d_B, *d_C;

    int a = A.rows();
    int b = A.cols();

    Eigen::MatrixXf Af = A.cast<float>();
    Eigen::MatrixXf Bf = B.cast<float>();
    Eigen::MatrixXf Cf = C.cast<float>();

    hipMalloc(&d_A,a*b*sizeof(float));
    hipMemcpy(d_A,Af.data(),a*b*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc(&d_B,a*b*sizeof(float));
    hipMemcpy(d_B,Bf.data(),a*b*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc(&d_C,a*b*sizeof(float));

    // declare the number of blocks per grid and the number of threads per
    // block
    dim3 dimBlock(32,32);
    dim3 dimGrid(b/dimBlock.x,a/dimBlock.y);

    matrixMultiplicationKernelEW<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,a,b);

    // Retrieve result and free data
    hipMemcpy(Cf.data(),d_C,a*b*sizeof(float),hipMemcpyDeviceToHost);

    C = Cf.cast<double>();

    hipDeviceSynchronize();
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void SimulateGPU::ewMD(const Eigen::MatrixXd& A, const Eigen::MatrixXd& B,
        Eigen::MatrixXd& C) {

    int device = 0;
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    maxMultiProcessors = properties.multiProcessorCount;
    maxThreadsPerBlock = properties.maxThreadsPerBlock;

    if ((A.cols() != B.cols()) || (A.rows() != B.rows())) {
        throw "SimulateGPU: matrixMultiplication: Matrix dimensions do not match!";
    }

    float *d_A, *d_B, *d_C;

    int a = A.rows();
    int b = A.cols();

    Eigen::MatrixXf Af = A.cast<float>();
    Eigen::MatrixXf Bf = B.cast<float>();
    Eigen::MatrixXf Cf = C.cast<float>();

    hipMalloc(&d_A,a*b*sizeof(float));
    hipMemcpy(d_A,Af.data(),a*b*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc(&d_B,a*b*sizeof(float));
    hipMemcpy(d_B,Bf.data(),a*b*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc(&d_C,a*b*sizeof(float));

    // declare the number of blocks per grid and the number of threads per
    // block
    dim3 dimBlock(32,32);
    dim3 dimGrid(b/dimBlock.x,a/dimBlock.y);

    matrixDivisionKernelEW<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,a,b);

    // Retrieve result and free data
    hipMemcpy(Cf.data(),d_C,a*b*sizeof(float),hipMemcpyDeviceToHost);

    C = Cf.cast<double>();

    hipDeviceSynchronize();
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void SimulateGPU::lineSegmentIntersect(const Eigen::MatrixXd& XY1, const
        Eigen::MatrixXd& XY2, Eigen::VectorXi& crossings) {

    int device = 0;
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    int maxThreadsPerBlock = properties.maxThreadsPerBlock;
    int maxBlocksPerGrid = 65536;

    // Precompute necessary vectors to be shared across threads
    Eigen::VectorXf X4_X3 = (XY2.block(0,2,XY2.rows(),1) -
            XY2.block(0,0,XY2.rows(),1)).cast<float>();
    Eigen::VectorXf Y4_Y3 = (XY2.block(0,3,XY2.rows(),1) -
            XY2.block(0,1,XY2.rows(),1)).cast<float>();
    Eigen::VectorXf X2_X1 = (XY1.block(0,2,XY1.rows(),1) -
            XY1.block(0,0,XY1.rows(),1)).cast<float>();
    Eigen::VectorXf Y2_Y1 = (XY1.block(0,3,XY1.rows(),1) -
            XY1.block(0,1,XY1.rows(),1)).cast<float>();

    Eigen::MatrixXf XY1f = XY1.cast<float>();
    Eigen::MatrixXf XY2f = XY2.cast<float>();

    // Allocate space on the GPU
    float *d_XY1, *d_XY2, *d_X4_X3, *d_Y4_Y3, *d_X2_X1, *d_Y2_Y1;
    int *d_adjacency, *d_cross;

    hipMalloc(&d_XY1,XY1.rows()*XY1.cols()*sizeof(float));
    hipMemcpy(d_XY1,XY1f.data(),XY1.rows()*XY1.cols()*sizeof(float),
            hipMemcpyHostToDevice);
    hipMalloc(&d_XY2,XY2.rows()*XY2.cols()*sizeof(float));
    hipMemcpy(d_XY2,XY2f.data(),XY2.rows()*XY2.cols()*sizeof(float),
            hipMemcpyHostToDevice);

    hipMalloc(&d_X4_X3,XY2.rows()*sizeof(float));
    hipMemcpy(d_X4_X3,X4_X3.data(),XY2.rows()*sizeof(float),
            hipMemcpyHostToDevice);
    hipMalloc(&d_Y4_Y3,XY2.rows()*sizeof(float));
    hipMemcpy(d_Y4_Y3,Y4_Y3.data(),XY2.rows()*sizeof(float),
            hipMemcpyHostToDevice);
    hipMalloc(&d_X2_X1,XY1.rows()*sizeof(float));
    hipMemcpy(d_X2_X1,X2_X1.data(),XY1.rows()*sizeof(float),
            hipMemcpyHostToDevice);
    hipMalloc(&d_Y2_Y1,XY1.rows()*sizeof(float));
    hipMemcpy(d_Y2_Y1,Y2_Y1.data(),XY1.rows()*sizeof(float),
            hipMemcpyHostToDevice);
    hipMalloc(&d_adjacency,XY1.rows()*XY2.rows()*sizeof(int));
    hipMalloc(&d_cross,XY1.rows()*sizeof(int));

    // Compute the road crossings for each transition
    int noCombos = XY1.rows()*XY2.rows();
    int noBlocks = (noCombos % maxThreadsPerBlock) ?
            (noCombos/maxThreadsPerBlock + 1) : (noCombos/maxThreadsPerBlock);
    double number = (double)(noBlocks)/(((double)maxBlocksPerGrid)*
            ((double)maxBlocksPerGrid));
    int blockYDim = ((number - floor(number)) > 0 ) ? (int)number + 1 :
            (int)number;
    int blockXDim = (int)min(maxBlocksPerGrid,noBlocks);

    dim3 dimGrid(blockXDim,blockYDim);
    pathAdjacencyKernel<<<dimGrid,maxThreadsPerBlock>>>(XY1.rows(),XY2.rows(),
            d_XY1,d_XY2,d_X4_X3,d_Y4_Y3,d_X2_X1,d_Y2_Y1,d_adjacency);
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
      fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
    }

    // Sum the number
    noBlocks = (XY1.rows() % maxThreadsPerBlock)? (int)(XY1.rows()/
            maxThreadsPerBlock + 1) : (int)(XY1.rows()/maxThreadsPerBlock);
    roadCrossingsKernel<<<noBlocks,maxThreadsPerBlock>>>(XY1.rows(),
            XY2.rows(),d_adjacency,d_cross);
    hipDeviceSynchronize();

    // Retrieve results
    hipMemcpy(crossings.data(),d_cross,XY1.rows()*sizeof(int),
            hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    // Free memory
    hipFree(d_X4_X3);
    hipFree(d_Y4_Y3);
    hipFree(d_X2_X1);
    hipFree(d_Y2_Y1);
    hipFree(d_cross);
}

void SimulateGPU::buildPatches(int W, int H, int skpx, int skpy, int xres,
        int yres, int noRegions, double xspacing, double yspacing, double
        subPatchArea, HabitatTypePtr habTyp, const Eigen::MatrixXi&
        labelledImage, const Eigen::MatrixXd& populations,
        std::vector<HabitatPatchPtr>& patches, double& initPop,
        Eigen::VectorXd& initPops, int& noPatches) {

    // Get device properties
    int device = 0;
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    maxMultiProcessors = properties.multiProcessorCount;
    maxThreadsPerBlock = properties.maxThreadsPerBlock;

    Eigen::MatrixXf popsFloat = populations.cast<float>();

    float *results, *d_results, *d_populations;
    int *d_labelledImage;

    results = (float*)malloc(xres*yres*noRegions*5*sizeof(float));
    hipMalloc((void **)&d_results,xres*yres*noRegions*5*sizeof(float));

    hipMalloc((void **)&d_labelledImage,H*W*sizeof(int));
    hipMemcpy(d_labelledImage,labelledImage.data(),H*W*sizeof(int),
            hipMemcpyHostToDevice);

    hipMalloc((void **)&d_populations,H*W*sizeof(float));
    hipMemcpy(d_populations,popsFloat.data(),H*W*sizeof(float),
            hipMemcpyHostToDevice);

    int noBlocks = ((xres*yres*noRegions) % maxThreadsPerBlock)? (int)(xres*
            yres*noRegions/maxThreadsPerBlock + 1) : (int)(xres*yres*noRegions/
            maxThreadsPerBlock);
    int noThreadsPerBlock = min(maxThreadsPerBlock,xres*yres*noRegions);

    patchComputation<<<noBlocks,noThreadsPerBlock>>>(xres*yres*noRegions,
            W, H, skpx, skpy, xres,yres,(float)subPatchArea,(float)xspacing,
            (float)yspacing,(float)habTyp->getMaxPop(),noRegions,
            d_labelledImage,d_populations,d_results);
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
      fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
    }

    hipMemcpy(results,d_results,xres*yres*noRegions*5*sizeof(float),
               hipMemcpyDeviceToHost);

    // Now turn the results into patches
    for (int ii = 0; ii < xres*yres*noRegions; ii++) {
        if (results[5*ii] > 0) {
            // Create new patch to add to patches vector
            HabitatPatchPtr hab(new HabitatPatch());
            hab->setArea((double)results[5*ii]);
            hab->setCX((double)results[ii+3]);
            hab->setCY((double)results[ii+4]);
            hab->setPopulation((double)results[5*ii+2]);
            hab->setCapacity((double)results[5*ii+1]);
            initPop += (double)results[5*ii];
            initPops(noPatches) = (double)results[5*ii];
            patches[noPatches++] = hab;
        }
    }

    hipFree(d_populations);
    hipFree(d_labelledImage);
    hipFree(d_results);
    free(results);
}

void SimulateGPU::simulateMTECUDA(SimulatorPtr sim,
        std::vector<SpeciesRoadPatchesPtr>& srp,
        std::vector<Eigen::VectorXd>& initPops,
        std::vector<Eigen::VectorXd>& capacities,
        Eigen::MatrixXd& endPops) {

    // Get device properties
    int device = 0;
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    maxMultiProcessors = properties.multiProcessorCount;
    maxThreadsPerBlock = properties.maxThreadsPerBlock;

    // We convert all inputs to floats from double as CUDA is much faster in
    // single precision than double precision

    // Get important values for computation
    int nYears = sim->getRoad()->getOptimiser()->getEconomic()->getYears();
    int noPaths = sim->getRoad()->getOptimiser()->getOtherInputs()->
            getNoPaths();

    // Get the important values for the road first and convert them to
    // formats that the kernel can use

    for (int ii = 0; ii < srp.size(); ii++) {

        // Species parameters
        double stepSize = sim->getRoad()->getOptimiser()->getEconomic()->
                getTimeStep();
        float grm = (float)(srp[ii]->getSpecies()->getGrowthRateMean()*
                stepSize);
        float grsd = (float)(srp[ii]->getSpecies()->getGrowthRateSD()*
                stepSize);
        int nPatches = capacities[ii].size();

        float *eps, *d_initPops, *d_eps, *d_caps, *d_mmm;

        // RANDOM MATRIX
        float *d_random_floats;
        hiprandGenerator_t gen;
        srand(time(NULL));
        int _seed = rand();
        //allocate space for 100 floats on the GPU
        //could also do this with thrust vectors and pass a raw pointer
        hipMalloc((void **)&d_random_floats, sizeof(float) *nYears*noPaths*
                nPatches);
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, _seed);
        hiprandGenerateNormal(gen, d_random_floats, nYears*noPaths*nPatches,
                0.0f,1.0f);
        hiprandDestroyGenerator(gen);
        hipDeviceSynchronize();

        // INITIAL POPULATIONS
        Eigen::MatrixXf initPopsF = initPops[ii].cast<float>();
        hipMalloc(&d_initPops,initPops[ii].size()*sizeof(float));
        hipMemcpy(d_initPops,initPopsF.data(),
                initPops[ii].size()*sizeof(float),hipMemcpyHostToDevice);

        // END POPULATIONS
        eps = (float*)malloc(noPaths*sizeof(float));
        hipMalloc(&d_eps, noPaths*sizeof(float));

        for (int jj = 0; jj < noPaths; jj++) {
            eps[jj] = 0.0f;
        }

        hipMemcpy(d_eps,eps,noPaths*sizeof(float),hipMemcpyHostToDevice);

        // CAPACITIES
        Eigen::VectorXf capsF = capacities[ii].cast<float>();
        hipMalloc(&d_caps,capacities[ii].size()*sizeof(float));
        hipMemcpy(d_caps,capsF.data(),capacities[ii].size()*sizeof(float),
                hipMemcpyHostToDevice);

        // MOVEMENT AND MORTALITY MATRIX
        // We use the highest flow rate in the vector of survival matrices
        const Eigen::MatrixXd& transProbs = srp[ii]->getTransProbs();
        const Eigen::MatrixXd& survProbs = srp[ii]->getSurvivalProbs()[
                srp[ii]->getSurvivalProbs().size()-1];
        Eigen::MatrixXf mmm = (transProbs.array()*survProbs.array()).
                cast<float>();

        hipMalloc(&d_mmm,mmm.rows()*mmm.cols()*sizeof(float));
        hipMemcpy(d_mmm,mmm.data(),mmm.rows()*mmm.cols()*
                sizeof(float),hipMemcpyHostToDevice);

        ///////////////////////////////////////////////////////////////////////
        // Perform N simulation paths. Currently, there is no species
        // interaction, so we run each kernel separately and do not need to use
        // the Thrust library.
        int noBlocks = (int)(noPaths % maxThreadsPerBlock)?
                (int)(noPaths/maxThreadsPerBlock + 1) :
                (int)(noPaths/maxThreadsPerBlock);
        int noThreadsPerBlock = min(noPaths,maxThreadsPerBlock);

        mteKernel<<<noBlocks,noThreadsPerBlock>>>
                (noPaths,nYears,capacities[ii].size(),grm,grsd,d_initPops,
                d_caps,d_mmm,d_eps,d_random_floats);
        hipDeviceSynchronize();

        // Retrieve results
        hipMemcpy(eps,d_eps,srp.size()*sizeof(float),hipMemcpyDeviceToHost);

        for (int jj = 0; jj < noPaths; jj++) {
            endPops(jj,ii) = eps[jj];
        }

        // Free memory
        hipDeviceSynchronize();
        hipFree(d_random_floats);
        hipFree(d_initPops);
        hipFree(d_eps);
        hipFree(d_caps);
        hipFree(d_mmm);
        free(eps);
    }
}

void SimulateGPU::simulateROVCUDA(SimulatorPtr sim,
        std::vector<SpeciesRoadPatchesPtr>& srp,
        std::vector<std::vector<Eigen::MatrixXd> > &aars,
        std::vector<Eigen::MatrixXd> &totalPops, Eigen::MatrixXd& condExp,
        Eigen::MatrixXi& optCont) {
    // Currently there is no species interaction. This can be a future question
    // and would be an interesting extension on how it can be implemented,
    // what the surrogate looks like and how the patches are formed.

    // Get device properties
    int device = 0;
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    maxMultiProcessors = properties.multiProcessorCount;
    maxThreadsPerBlock = properties.maxThreadsPerBlock;

    // Get general properties
    OptimiserPtr optimiser = sim->getRoad()->getOptimiser();
    ExperimentalScenarioPtr scenario = optimiser->getScenario();
    VariableParametersPtr varParams = optimiser->getVariableParams();
    int sc = scenario->getProgram();
    TrafficProgramPtr program = optimiser->getPrograms()[sc];
    std::vector<CommodityPtr> commodities = optimiser->getEconomic()->
            getCommodities();
    std::vector<CommodityPtr> fuels = optimiser->getEconomic()->
            getFuels();

    // Get important values for computation
    int nYears = sim->getRoad()->getOptimiser()->getEconomic()->getYears();
    int noPaths = sim->getRoad()->getOptimiser()->getOtherInputs()->
            getNoPaths();
    int noControls = program->getFlowRates().size();
    int noUncertainties = commodities.size() + fuels.size();

    double unitProfit = sim->getRoad()->getAttributes()->getUnitVarRevenue();
    double unitCost = sim->getRoad()->getAttributes()->getUnitVarCosts();
    double stepSize = optimiser->getEconomic()->getTimeStep();

    // Get the important values for the road first and convert them to formats
    // that the kernel can use

    // Initialise CUDA memory /////////////////////////////////////////////////

    // 1. Transition and survival matrices for each species and each control
    float *transitions, *survival, *initPops, *capacities, *speciesParams,
            *uncertParams, *d_transitions, *d_survival, *d_initPops,
            *d_tempPops, *d_capacities, *d_speciesParams, *d_uncertParams;

    int *noPatches, *d_noPatches;

    noPatches = (int*)malloc(srp.size()*sizeof(int));

    int patches = 0;
    int transition = 0;

    for (int ii = 0; ii < srp.size(); ii++) {
        noPatches[ii] = srp[ii]->getHabPatches().size();
        patches += noPatches[ii];
        transition += pow(patches,2);
    }

    initPops = (float*)malloc(patches*sizeof(float));
    capacities = (float*)malloc(patches*sizeof(float));
    transitions = (float*)malloc(transition*sizeof(float));
    survival = (float*)malloc(transition*noControls*sizeof(float));
    speciesParams = (float*)malloc(srp.size()*2*sizeof(float));
    uncertParams = (float*)malloc(noUncertainties*6*sizeof(float));

    hipMalloc((void**)&d_noPatches,srp.size*sizeof(int));
    hipMalloc((void**)&d_initPops,patches*sizeof(float));
    hipMalloc((void**)&d_capacities,patches*sizeof(float));
    hipMalloc((void**)&d_transitions,transition*sizeof(float));
    hipMalloc((void**)&d_survival,transition*noControls*sizeof(float));
    hipMalloc((void**)&d_speciesParams,srp.size()*2*sizeof(float));
    hipMalloc((void**)&d_uncertParams,(noUncertainties*6*sizeof(float));

    int counter1 = 0;
    int counter2 = 0;
    int counter3 = 0;

    // Read in the information into the correct format
    for (int ii = 0; ii < srp.size(); ii++) {
        memcpy(initPops+counter1,srp[ii]->getInitPops().data(),
                srp[ii]->getHabPatches().size());
        memcpy(capacities+counter1,srp[ii]->getCapacities().data(),
                srp[ii]->getHabPatches().size());

        speciesParams[counter1] = srp[ii]->getSpecies()->getGrowthRateMean()*
                varParams->getGrowthRatesMultipliers()(scenario->getPopGR());
        speciesParams[counter1+1] = srp[ii]->getSpecies()->getGrowthRateSD()*
                varParams->getGrowthRateSDMultipliers()(scenario->getPopGRSD());

        counter1 += srp[ii]->getHabPatches().size();

        memcpy(transitions+counter2,srp[ii]->getTransProbs().data(),
                pow(srp[ii]->getHabPatches().size(),2));
        counter2 += pow(srp[ii]->getHabPatches().size(),2);

        for (int jj = 0; jj < noControls; jj++) {
            memcpy(survival+counter3,srp[ii]->getSurvivalProbs()[jj].data(),
                pow(srp[ii]->getHabPatches().size(),2));
            counter3 += pow(srp[ii]->getHabPatches().size(),2);
        }
    }

    for (int ii = 0; ii < fuels.size()); ii++) {
        uncertParams[noUncertainties*ii] = fuels[ii]->getCurrent();
        uncertParams[noUncertainties*ii+1] = fuels[ii]->getMean();
        uncertParams[noUncertainties*ii+2] = fuels[ii]->getNoiseSD();
        uncertParams[noUncertainties*ii+3] = fuels[ii]->getMRStrength();
        uncertParams[noUncertainties*ii+4] = fuels[ii]->getPoissonJump();
        uncertParams[noUncertainties*ii+5] = fuels[ii]->getJumpProb();
    }

    for (int ii = 0; ii < commodities.size()); ii++) {
        uncertParams[fuels.size()*6 + noUncertainties*ii] =
                commodities[ii]->getCurrent();
        uncertParams[fuels.size()*6 + noUncertainties*ii+1] =
                commodities[ii]->getMean();
        uncertParams[fuels.size()*6 + noUncertainties*ii+2] =
                commodities[ii]->getNoiseSD();
        uncertParams[fuels.size()*6 + noUncertainties*ii+3] =
                commodities[ii]->getMRStrength();
        uncertParams[fuels.size()*6 + noUncertainties*ii+4] =
                commodities[ii]->getPoissonJump();
        uncertParams[fuels.size()*6 + noUncertainties*ii+5] =
                commodities[ii]->getJumpProb();
    }

    // Transfer the data to the device
    hipMemcpy(d_noPatches,noPatches,srp.size()*sizeof(int));
    hipMemcpy(d_initPops,initPops,patches*sizeof(float),
            hipMemcpyHostToDevice);
    hipMemcpy(d_transitions,transitions,transition*sizeof(float),
            hipMemcpyHostToDevice);
    hipMemcpy(d_survival,survival,transition*sizeof(float),
            hipMemcpyHostToDevice);
    hipMemcpy(d_speciesParams,speciesParams,srp.size()*2*sizeof(float));
    hipMemcpy(d_uncertParams,uncertParams,noUncertainties*6*sizeof(float));

    float *d_randCont, *d_growthRate, *d_uBrownian, *d_uJumpSizes,
            *d_uJumps, *d_uResults;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, _seed);

    // 2. Random matrices for randomised control
    hiprandGenerateUniform(gen, d_randCont, nYears*noPaths);

    // 3. Random matrices for growth rate parameter for species
    hiprandGenerateNormal(gen, d_growthRate, nYears*noPaths*patches*srp.size(),
            0.0f,1.0f);

    // 4. Random matrices for other uncertainties
    hiprandGenerateNormal(gen, d_uBrownian, nYears*noPaths*noUncertainties,0.0f,
            1.0f);

    hiprandGenerateNormal(gen, d_uJumpSizes, nYears*noPaths*noUncertainties,
            0.0f,1.0f);

    hiprandGenerateUniform(gen, d_uJumps, nYears*noPaths*noUncertainties);

    // Destroy generator
    hiprandDestroyGenerator(gen);
    hipDeviceSynchronize();

    // Finally, allocate space on the device for the path results. This is what
    // we use in our policy map.
    float *d_totalPops, *d_aars, *d_mcPops;
    hipMalloc(&d_totalPops,(nYears+1)*noPaths*sizeof(float));
    hipMalloc(&d_aars,(nYears+1)*noPaths*noControls*sizeof(float));
    hipMalloc(&d_mcPops,(nYears+1)*noPaths*patches*sizeof(float));

    // Compute forward paths (CUDA kernel)
    int noBlocks = (int)(noPaths % maxThreadsPerBlock) ?
            (int)(noPaths/maxThreadsPerBlock + 1) :
            (int)(noPaths/maxThreadsPerBlock);
    int noThreadsPerBlock = min(noPaths,maxThreadsPerBlock);

    forwardPathKernel<<<noBlocks,noThreadsPerBlock>>>();
    hipDeviceSynchronize();

    // Choose the appropriate method
    switch (optimiser->getROVMethod()) {

        case Optimiser::ALGO1:
        {
        }
        break;

        case Optimiser::ALGO2:
        {
        }
        break;

        case Optimiser::ALGO3:
        {
        }
        break;

        case Optimiser::ALGO4:
        {
        }
        break;

        case Optimiser::ALGO5:
        {
        }
        break;

        case Optimiser::ALGO6:
        {

        }
        break;

        case Optimiser::ALGO7:
        {
        }
        break;

        default:
        {
        }
        break;
    }
}
